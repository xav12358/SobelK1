#include <hip/hip_runtime.h>

#include <stdio.h>

__global__ void Pass0_clean(int32_t *ptBufferDxOut,
                            int32_t *ptBufferDyOut,
                            u_int32_t *ptSobelOut,
                            u_int32_t *ptLabelOut,
                            u_int32_t *ptArea,
                            u_int32_t *ptOut,
                            u_int32_t *u32_BufferCornerList_Device,
                            int w,int h)
{


    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;
    int xglobal = x+threadIdx.x;
    int yglobal = y+threadIdx.y;

    if(xglobal>w || yglobal >h)
    {
        return;
    }

    ptBufferDxOut[xglobal+yglobal*w]    = 0;
    ptBufferDyOut[xglobal+yglobal*w]    = 0;
    ptSobelOut[xglobal+yglobal*w]       = 0;
    ptLabelOut[xglobal+yglobal*w]       = 0;
    ptArea[xglobal+yglobal*w]           = 0;
    ptOut[xglobal+yglobal*w]            = 0;
    u32_BufferCornerList_Device[xglobal+yglobal*w] = 0;
}
