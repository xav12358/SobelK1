#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>


#include "global_var.h"
#include "markerdetector.h"

texture<unsigned char, 2,hipReadModeNormalizedFloat> Image_Gray;


__global__ void ExtractMarker56p( u_int8_t *ptTemplateMarker)
{

    for(int i=threadIdx.x;i<_MARKER_MAX_SIZE;i+=blockDim.x)
    {
        for(int j=threadIdx.y;j<_MARKER_MAX_SIZE;j+=blockDim.y)
        {
            float x00 = (float)i;
            float y00 = (float)j;
                      u_int8_t u8_Val = 255* tex2D(Image_Gray,x00+0.5,y00+0.5);
            ptTemplateMarker[blockIdx.x*_MARKER_MAX_SIZE*_MARKER_MAX_SIZE + (i + j*_MARKER_MAX_SIZE) ] = u8_Val;
        }
    }

}



