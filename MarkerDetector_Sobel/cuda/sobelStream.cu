#include "hip/hip_runtime.h"


#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "sobelStream.h"

//#include "hip/hip_vector_types.h"
#include <android/log.h>

#define APP_NAME "CUDA_CORE"

#define LOGD(...) ((void)__android_log_print(ANDROID_LOG_DEBUG,  \
                                             APP_NAME, \
                                             __VA_ARGS__))



#define checkCudaErrors(err)           __checkCudaErrors (err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err)
    {
        LOGD("CUDA Runtime API error: %s", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// This will output the proper error string when calling hipGetLastError
#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)

inline void __getLastCudaError(const char *errorMessage, const char *file, const int line)
{
    hipError_t err = hipGetLastError();

    if (hipSuccess != err)
    {
        LOGD("CUDA error %s:%s", errorMessage, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


#define WIDTH   640
#define HEIGHT  480
#define NB_STREAM 10

#define BLOC_X 32
#define BLOC_Y 32

hipStream_t *Stream;

hipArray * Array_PatchsMaxDevice;
texture<u_int8_t, 2,hipReadModeElementType> Image;


__global__ void SobelKernelSTREAM(u_int8_t *ptDataIn ,u_int8_t *ptDataOut,int hoffset,int widthToProcess,int heightToProcess)
{

    __shared__ u_int8_t localImage[WIDTH*HEIGHT/10];
    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;


    for(int i=threadIdx.x;i<widthToProcess;i+=blockDim.x)
    {
        for(int j=threadIdx.y;j<heightToProcess;j+=blockDim.y)
        {
            localImage[i+j*WIDTH] = ptDataIn[i +(j+hoffset)*WIDTH];
        }

    }

    syncthreads();

    int indiceKernelx[8] = {-1,0,1,1,1,0,-1,-1};
    int indiceKernely[8] = {-1,-1,-1,0,1,1,1,0};

    int indiceKernelWeightX[8] = {-1,0,1,2,1,0,-1,-2};
    int indiceKernelWeightY[8] = {-1,-2,-1,0,1,2,1,0};

    int x0,y0,c;
    int xout,yout;
    for(int i=threadIdx.x;i<widthToProcess;i+=blockDim.x)
    {
        for(int j=threadIdx.y;j<heightToProcess;j+=blockDim.y)
        {
            bool isValid = true;
            for(int k=0;k<8;k++)
            {

                x0 = indiceKernelx[k]+1;
                y0 = indiceKernely[k]+1;
                if(x0<0 || y0<0 || x0<widthToProcess || y0>heightToProcess)
                    break;
                c =  localImage[i+x0 +(j+y0)*WIDTH];

                xout += c*indiceKernelWeightX[k];
                yout += c*indiceKernelWeightY[k];
            }
            if(!isValid)
                continue;
            ptDataOut[i+(j+hoffset)*WIDTH] = xout*xout;//sqrt(xout*xout+yout*yout) ;
        }
    }
}

#define BLOCK_SIZE_X 32
#define BLOCK_SIZE_Y 32

__global__ void SobelKernel(u_int8_t *ptDataIn ,u_int8_t *ptDataOut,int w,int h)
{

    int indiceKernelx[8] = {-1,0,1,1,1,0,-1,-1};
    int indiceKernely[8] = {-1,-1,-1,0,1,1,1,0};

    int indiceKernelWeightX[8] = {-1,0,1,2,1,0,-1,-2};
    int indiceKernelWeightY[8] = {-1,-2,-1,0,1,2,1,0};

    __shared__ u_int8_t  LocalMem[(BLOCK_SIZE_X+2)*(BLOCK_SIZE_Y+2)];

    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;
    int xglobal = x+threadIdx.x;
    int yglobal = y+threadIdx.y;


    int xout =0;
    int yout =0;
    int c = 0;
    int x0 = 0,y0 = 0;


    for(int i=threadIdx.x;i<(BLOCK_SIZE_X+2) && (x+i)<w;i=i+blockDim.x)
    {
        for(int j=threadIdx.y;j<(BLOCK_SIZE_Y+2) && (y+j)<h;j=j+blockDim.y)
        {
            if((x+i)<0 || (y+j)<0) continue;
            LocalMem[i+j*(BLOCK_SIZE_Y+2)] = ptDataIn[x+i + (y+ j)*w];
        }
    }


    if(xglobal < 1 || xglobal >= 640-1 || yglobal < 1 || yglobal >= 480-1)
    {
        ptDataOut[xglobal+yglobal*w] = 0;
        return;
    }


    syncthreads();

    for(int i=0;i<8;i++)
    {
        x0 = indiceKernelx[i]+1;
        y0 = indiceKernely[i]+1;
        c =  LocalMem[threadIdx.x+x0 +(threadIdx.y+y0)*(BLOCK_SIZE_Y+2)];

        xout += c*indiceKernelWeightX[i];
        yout += c*indiceKernelWeightY[i];
    }

    ptDataOut[xglobal+yglobal*w]       = xout*xout + yout*yout;


}

__global__ void ClearKernel(u_int8_t *ptDataOut,int w,int h)
{


    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;

    int xglobal = x + threadIdx.x;
    int yglobal = y + threadIdx.y;


    if(xglobal>=w || yglobal >= h  )
        return;


    ptDataOut[xglobal +yglobal*WIDTH] = 0;

}



void processFilter(void)
{

    u_int8_t *u8_PtImageHost;
    u_int8_t *u8_PtImageDevice;

    u_int8_t *u8_ptDataOutHost;
    u_int8_t *u8_ptDataOutDevice;

    u_int8_t u8_Used[NB_STREAM];

    //    u8_ptDataOutHost = (u_int8_t *)malloc(WIDTH*HEIGHT*sizeof(u_int8_t));
    //    u8_PtImageHost = (u_int8_t *)malloc(WIDTH*HEIGHT*sizeof(u_int8_t));

    checkCudaErrors(hipHostAlloc((void**)&u8_ptDataOutHost,WIDTH*HEIGHT*sizeof(u_int8_t),hipHostMallocDefault));
    checkCudaErrors(hipHostAlloc((void**)&u8_PtImageHost,WIDTH*HEIGHT*sizeof(u_int8_t),hipHostMallocDefault));


    checkCudaErrors(hipMalloc((void**)&u8_ptDataOutDevice,WIDTH*HEIGHT*sizeof(u_int8_t)));
    checkCudaErrors(hipMalloc((void**)&u8_PtImageDevice,WIDTH*HEIGHT*sizeof(u_int8_t)));

    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    checkCudaErrors(hipMallocArray(&Array_PatchsMaxDevice, &channelDesc,WIDTH,HEIGHT ));
    checkCudaErrors(hipBindTextureToArray(Image,Array_PatchsMaxDevice));


    dim3 threads(BLOC_X,BLOC_Y);
    dim3 blocks(1,1);
    //    ClearKernel<<<blocks,threads>>>(u8_ptDataOutDevice,WIDTH,HEIGHT);


    int blockh = HEIGHT/NB_STREAM;


    Stream = (hipStream_t *) malloc(NB_STREAM * sizeof(hipStream_t));

    for (int i = 0; i < NB_STREAM; i++)
    {
        checkCudaErrors(hipStreamCreate(&(Stream[i])));
    }

    int eventflags = hipEventDefault;

    hipEvent_t Start;
    hipEvent_t Stop;
    checkCudaErrors(hipEventCreateWithFlags(&Start, eventflags));
    checkCudaErrors(hipEventCreateWithFlags(&Stop, eventflags));

    hipEventRecord(Start, 0);

    /////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////
    for(int i=0;i<NB_STREAM;i++)
    {
        if(i == 0)
        {
            int localHeight  = blockh;
            checkCudaErrors(hipMemcpyAsync(u8_PtImageHost,u8_PtImageDevice,WIDTH*(localHeight-1),hipMemcpyHostToDevice,Stream[i]));
            SobelKernelSTREAM<<<blocks,threads,0,Stream[i]>>>(u8_PtImageDevice,u8_ptDataOutDevice,0,WIDTH,localHeight-1);
            checkCudaErrors(hipMemcpyAsync(u8_ptDataOutHost,u8_ptDataOutDevice,WIDTH*(localHeight-1)*sizeof(u_int8_t),hipMemcpyDeviceToHost,Stream[i]));

            u8_Used[i] = 1;

        }else{

            int ioffsetImage =  WIDTH*(HEIGHT/NB_STREAM  );
            int hoffset = HEIGHT/NB_STREAM *i;
            int hoffsetkernel = HEIGHT/NB_STREAM -1 + HEIGHT/NB_STREAM* (i-1);
            int localHeight  = min(HEIGHT - (blockh*i),blockh);

            printf("hoffset %d localHeight %d\n",hoffset,localHeight);
            checkCudaErrors(hipMemcpyAsync(&u8_PtImageHost[hoffset*WIDTH],&u8_PtImageDevice[hoffset*WIDTH],WIDTH*(localHeight-1),hipMemcpyHostToDevice,Stream[i]));
            SobelKernelSTREAM<<<blocks,threads,0,Stream[i]>>>(u8_PtImageDevice,u8_ptDataOutDevice,hoffset,WIDTH,localHeight);
            checkCudaErrors(hipGetLastError());
            checkCudaErrors(hipMemcpyAsync(&u8_ptDataOutHost[hoffsetkernel*WIDTH],&u8_ptDataOutDevice[hoffsetkernel*WIDTH],WIDTH*localHeight*sizeof(u_int8_t),hipMemcpyDeviceToHost,Stream[i]));

            u8_Used[i] = 1;
            if(HEIGHT - (blockh +1 +blockh*(i-1))<=0)
            {
                break;
            }
        }
    }

    float dt_msK;
    hipEventRecord(Stop, 0);
    hipEventSynchronize(Start);
    hipEventSynchronize(Stop);
    hipEventElapsedTime(&dt_msK, Start, Stop);

//    LOGD("WITH STREAM dt_ms %f \n",dt_ms);

    /////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////

    float dt_ms[3];

    dim3 threadsK(BLOCK_SIZE_X,BLOCK_SIZE_Y);
    dim3 blocksK(ceil((float)WIDTH/BLOC_X),ceil((float)HEIGHT/BLOC_Y));

    hipEventRecord(Start, 0);
    checkCudaErrors(hipMemcpy(u8_PtImageHost,u8_PtImageDevice,WIDTH*HEIGHT,hipMemcpyHostToDevice));
    hipEventRecord(Stop, 0);
    hipEventSynchronize(Start);
    hipEventSynchronize(Stop);
    hipEventElapsedTime(&dt_ms[0], Start, Stop);


    hipEventRecord(Start, 0);
    SobelKernel<<<blocksK,threadsK>>>(u8_PtImageDevice,u8_ptDataOutDevice,WIDTH,HEIGHT);
    hipEventRecord(Stop, 0);
    hipEventSynchronize(Start);
    hipEventSynchronize(Stop);
    hipEventElapsedTime(&dt_ms[1], Start, Stop);

    hipEventRecord(Start, 0);
    checkCudaErrors(hipMemcpy(u8_ptDataOutHost,u8_ptDataOutDevice,WIDTH*(HEIGHT)*sizeof(u_int8_t),hipMemcpyDeviceToHost));
    hipEventRecord(Stop, 0);
    hipEventSynchronize(Start);
    hipEventSynchronize(Stop);
    hipEventElapsedTime(&dt_ms[2], Start, Stop);

    LOGD("dt;%f;%f;%f\n",dt_ms[0],dt_ms[1],dt_ms[2]);

}

