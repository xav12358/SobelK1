#include "hip/hip_runtime.h"
#include "math/interpolation_func.h"
#include "markerdetector.h"


#include "cleanBuffers.cu"
#include "processSobel.cu"

#include <iostream>
#include <math.h>
#include <string>
#include <stdio.h>
#include <stdlib.h>



char *bufferToDisplay;

////////////////////////////
/// \brief MarkerDetector_gpu::MarkerDetector_gpu
/// \param rows
/// \param cols
///
MarkerDetector_gpu::MarkerDetector_gpu(int rows,int cols)

{

	bufferToDisplay = (char *)malloc(500*sizeof(char));

	LOGD("MarkerDetector_gpu0\n");
	if(VERBOSE)
		std::cout << "- MarkerDetector_gpu contructeur "<< std::endl;


	icols = cols;
	irows = rows;
	threads = dim3(BLOCK_SIZE_X,BLOCK_SIZE_Y);
	blocks  = dim3(ceil(cols/BLOCK_SIZE_X),ceil(rows/BLOCK_SIZE_Y));

	checkCudaErrors(hipMalloc((void **)&u8_GrayImage_Device,rows*cols*sizeof(u_int8_t)));
	checkCudaErrors(hipMalloc((void **)&u32_Buffer0_Device,rows*cols*sizeof(u_int32_t)));

	checkCudaErrors(hipMalloc((void **)&i32_BufferDx_Device,rows*cols*sizeof(int32_t)));
	checkCudaErrors(hipMalloc((void **)&i32_BufferDy_Device,rows*cols*sizeof(int32_t)));

	checkCudaErrors(hipMalloc((void **)&u32_SobelImage_Device,rows*cols*sizeof(u_int32_t)));
	checkCudaErrors(hipMalloc((void **)&u32_BufferArea_Device,rows*cols*sizeof(u_int32_t)));

	checkCudaErrors(hipMalloc((void **)&u32_Buffer5_Device,rows*cols*sizeof(u_int32_t)));
	checkCudaErrors(hipMalloc((void **)&u32_BufferCornerList_Device,rows*cols*sizeof(u_int32_t)));

	checkCudaErrors(hipMalloc((void**)&u8_extractedMarkers56p_Device,_MARKER_MAX_SIZE*_MARKER_MAX_SIZE*_MAX_NODE*sizeof(u_int8_t)));
	checkCudaErrors(hipMalloc((void**)&u8_extractedMarkers7p_Device,_MARKER_SMALL_SIZE*_MARKER_SMALL_SIZE*_MAX_NODE*sizeof(u_int8_t)));

	checkCudaErrors(hipMalloc((void**)&tranformationList_Device,_MAX_NODE*sizeof(TransformationM)));
	checkCudaErrors(hipMalloc((void**)&u8_MarkerListToSearch5p_Device,1024*5*5*sizeof(u_int8_t)));

	checkCudaErrors(hipMalloc((void**)&u16_DetectedId_Device,_MAX_NODE*sizeof(u_int16_t)));
	checkCudaErrors(hipMalloc((void**)&f2_SquareList_Device,4*_MAX_NODE*sizeof(float2)));

	checkCudaErrors(hipMalloc((void**)&f_tmp_AMatrix_Device,_MAX_NODE*8*8*sizeof(float)));


	/////////////////////////////////
	//    To Do: use hipHostAlloc
	u32_BufferCornerList_Host   = (u_int32_t*)malloc(rows*cols*sizeof(u_int32_t));
	u8_extractedMarkers56p_Host = (u_int8_t *)malloc(_MARKER_MAX_SIZE*_MARKER_MAX_SIZE*_MAX_NODE*sizeof(u_int8_t));

	u8_extractedMarkers7p_Host  = (u_int8_t *)malloc(_MARKER_SMALL_SIZE*_MARKER_SMALL_SIZE*_MAX_NODE*sizeof(u_int8_t));
	tranformationList_Host      = (TransformationM *)malloc(_MAX_NODE*sizeof(TransformationM));

	u8_MarkerListToSearch5p_Host= (u_int8_t *)malloc(1024*5*5*sizeof(u_int8_t));
	u16_DetectedId_Host         = (u_int16_t*)malloc(_MAX_NODE*sizeof(u_int16_t));
	f2_SquareList_Host          = (float2*)malloc(4*_MAX_NODE*sizeof(float2));


	// Create the list of 1024 markers
	createMarkerList();



//	Image_Gray.filterMode = hipFilterModeLinear;
//	Image_Gray.normalized = false;
//	hipChannelFormatDesc  desc = hipCreateChannelDesc<unsigned char>();
//	checkCudaErrors(hipBindTexture2D(0,&Image_Gray,u8_GrayImage_Device, &desc, icols , irows, icols));




	hipEventCreate(&Start,0);
	hipEventCreate(&Stop,0);

	//pFile = fopen ("myfile.csv","a");
	LOGD("SOBEL;CCL;Blobs;Corners;CornersGPU;Refine;ExtractMarker;DetectId\n");


}

/////////////////////////////////////
/// \brief MarkerDetector_gpu::~MarkerDetector_gpu
///
MarkerDetector_gpu::~MarkerDetector_gpu()
{
	// free device memory
	checkCudaErrors(hipFree(u8_GrayImage_Device));
	checkCudaErrors(hipFree(u32_Buffer0_Device));
	checkCudaErrors(hipFree(i32_BufferDx_Device));
	checkCudaErrors(hipFree(i32_BufferDy_Device));
	checkCudaErrors(hipFree(u32_SobelImage_Device));
	checkCudaErrors(hipFree(u32_BufferArea_Device));
	checkCudaErrors(hipFree(u32_Buffer5_Device));
	checkCudaErrors(hipFree(u32_BufferCornerList_Device));
	checkCudaErrors(hipFree(u8_extractedMarkers56p_Device));
	checkCudaErrors(hipFree(u8_extractedMarkers7p_Device));
	checkCudaErrors(hipFree(u8_MarkerListToSearch5p_Host));
	checkCudaErrors(hipFree(tranformationList_Device));

}






/////////////////////////////////
/// \brief MarkerDetector_gpu::processSobel
///
void MarkerDetector_gpu::processSobel(void)
{

	if(VERBOSE)
		std::cout << "- Process sobel filter " << std::endl;


	hipEventRecord(Start,0);
	Pass0_0<<<blocks,threads>>>(u8_GrayImage_Device,
			i32_BufferDx_Device,
			i32_BufferDy_Device,
			u32_SobelImage_Device,
			icols,irows);


	Pass0_1<<<blocks,threads>>>(i32_BufferDx_Device,
			i32_BufferDy_Device,
			u32_SobelImage_Device,
			icols,irows);


	Pass0_2<<<blocks,threads>>>(u32_SobelImage_Device,
			u32_Buffer0_Device,
			icols,irows);


	hipEventRecord(Stop, 0);
	hipEventSynchronize(Start);
	hipEventSynchronize(Stop);
	hipEventElapsedTime(&dt_ms[0], Start, Stop);

#if DEBUGG == 12
		cv::Mat gray_Label,adjMap_Label;
gray_Label.create(480,640,CV_32S);
double min,max;

checkCudaErrors(hipMemcpy(gray_Label.data ,u32_Buffer0_Device,icols*irows*sizeof(u_int32_t) ,hipMemcpyDeviceToHost));

cv::minMaxIdx(gray_Label, &min, &max);
cv::convertScaleAbs(gray_Label, adjMap_Label, 255 / max);
cv::imshow("gray_Label", adjMap_Label);
cv::waitKey(-1);
#endif

}


///////////////////////
/// \brief MarkerDetector_gpu::run
/// \param ptSrcHost
/// \param rows
/// \param cols
/// \param ipass
///
char * MarkerDetector_gpu::run(u_int8_t *ptSrcHost,int rows,int cols,int ipass)
{
	////////////////////////////
	// Clean all buffers
	std::cout << "- Clean buffers " << std::endl;
	Pass0_clean<<<blocks,threads>>>( i32_BufferDx_Device,
			i32_BufferDy_Device,
			u32_SobelImage_Device,
			u32_Buffer0_Device,
			u32_BufferArea_Device,
			u32_Buffer5_Device,
			u32_BufferCornerList_Device,
			icols,irows);

	///////////////////////////////////
	// Copy to the device memory
	std::cout << "- Copy to the device memory " << std::endl;
	checkCudaErrors(hipMemcpy(u8_GrayImage_Device,ptSrcHost,rows*cols*sizeof(u_int8_t),hipMemcpyHostToDevice));


	///////////////////////////////////////
	// Process sobel filter
	processSobel();



	 LOGD( "%.5f;%.5f;%.5f;%.5f;%.5f;%.5f;%.5f;%.5f;%d\n", dt_ms[0], dt_ms[1], dt_ms[2], dt_ms[3], dt_ms[4], dt_ms[5], dt_ms[6], dt_ms[7],SquareList.size());



	int cx = snprintf(bufferToDisplay, 500, "SOBEL CCL %.5f\nBlobs %.5f\nCorners %.5f\nCornersGPU %.5f\nRefine %.5f\nExtractMarker %.5f\nDetectId %.5f\n ID detected:", dt_ms[0], dt_ms[1], dt_ms[2], dt_ms[3], dt_ms[4], dt_ms[5], dt_ms[6], dt_ms[7] );

	for(int ind=0;ind<u16_MaxFeatureToProcess;ind++)
	{
		if(u16_DetectedId_Host[ind] != _MARKER_ID_INVALID )
		{
			cx = snprintf(bufferToDisplay +cx, 500, " %d ", SquareList[ind].GetID());
		}

	}

	return bufferToDisplay;
}
