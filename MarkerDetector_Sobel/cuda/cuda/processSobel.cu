#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "global_var.h"

__global__ void Pass0_0(u_int8_t *u8_GrayImageIn,
                        int32_t *ptBufferDxOut,
                        int32_t *ptBufferDyOut,
                        u_int32_t *ptSobelOut,
                        int w,int h)
{

    int indiceKernelx[8] = {-1,0,1,1,1,0,-1,-1};
    int indiceKernely[8] = {-1,-1,-1,0,1,1,1,0};

    int indiceKernelWeightX[8] = {-1,0,1,2,1,0,-1,-2};
    int indiceKernelWeightY[8] = {-1,-2,-1,0,1,2,1,0};

    __shared__ u_int8_t  LocalMem[(BLOCK_SIZE_X+2)*(BLOCK_SIZE_Y+2)];

    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;
    int xglobal = x+threadIdx.x;
    int yglobal = y+threadIdx.y;
    int xlocal  = threadIdx.x;
    int ylocal  = threadIdx.y;

    int xout =0;
    int yout =0;
    int c = 0;
    int x0 = 0,y0 = 0;


    for(int i=threadIdx.x;i<(BLOCK_SIZE_X+2) && (x+i)<w;i=i+blockDim.x)
    {
        for(int j=threadIdx.y;j<(BLOCK_SIZE_Y+2) && (y+j)<h;j=j+blockDim.y)
        {
            if((x+i)<0 || (y+j)<0) continue;
            LocalMem[i+j*(BLOCK_SIZE_Y+2)] = u8_GrayImageIn[x+i + (y+ j)*w];
        }
    }


    if(xglobal < 1 || xglobal >= 640-1 || yglobal < 1 || yglobal >= 480-1)
    {
        ptBufferDxOut[xglobal+yglobal*w] = -1;
        ptBufferDyOut[xglobal+yglobal*w] = -1;
        ptSobelOut[xglobal+yglobal*w] = 0;
        return;
    }


    syncthreads();

    for(int i=0;i<8;i++)
    {
        x0 = indiceKernelx[i]+1;
        y0 = indiceKernely[i]+1;
        c =  LocalMem[threadIdx.x+x0 +(threadIdx.y+y0)*(BLOCK_SIZE_Y+2)];

        xout += c*indiceKernelWeightX[i];
        yout += c*indiceKernelWeightY[i];
    }

    ptBufferDxOut[xglobal+yglobal*w]    = xout;
    ptBufferDyOut[xglobal+yglobal*w]    = yout;
    ptSobelOut[xglobal+yglobal*w]       = xout*xout + yout*yout;

}


__global__ void Pass0_1(int32_t *ptBufferDxIn,
                        int32_t *ptBufferDyIn,
                        u_int32_t *ptSobelOut,
                        int w,int h)
{
    __shared__ u_int32_t  LocalMemI[(BLOCK_SIZE_X+2)*(BLOCK_SIZE_Y+2)];
    __shared__ int32_t  LocalMemIx[(BLOCK_SIZE_X+2)*(BLOCK_SIZE_Y+2)];
    __shared__ int32_t  LocalMemIy[(BLOCK_SIZE_X+2)*(BLOCK_SIZE_Y+2)];

    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;
    int xglobal = x+threadIdx.x;
    int yglobal = y+threadIdx.y;
    int xlocal  = threadIdx.x;
    int ylocal  = threadIdx.y;

    for(int i=threadIdx.x;i<BLOCK_SIZE_X+2 && (x+i)<w;i=i+blockDim.x)
    {
        for(int j=threadIdx.y;j<BLOCK_SIZE_Y+2 && (y+j)<h;j=j+blockDim.y)
        {
            if((x+i)<0 || (y+j)<0) continue;
            LocalMemI[i+j*(BLOCK_SIZE_X+2)]  = ptSobelOut[x+i-1 + (y+j-1)*w];
            LocalMemIx[i+j*(BLOCK_SIZE_X+2)] = ptBufferDxIn[x+i-1 + (y+j-1)*w];
            LocalMemIy[i+j*(BLOCK_SIZE_X+2)] = ptBufferDyIn[x+i-1 + (y+j-1)*w];
        }
    }

    if(xglobal < 1 || xglobal >= 640-1 || yglobal < 1 || yglobal >= 480-1)
    {
        ptBufferDxIn[xglobal+yglobal*w] = -1;
        ptBufferDyIn[xglobal+yglobal*w] = -1;
        ptSobelOut[xglobal+yglobal*w] = -1;
        return;
    }


    syncthreads();

    int p0_local = (ylocal+1) * (BLOCK_SIZE_X+2) + xlocal+1;

    int x2 = 1, y2 = 1;
    int sx = LocalMemIx[p0_local], sy = LocalMemIy[p0_local];

    if (sx < 0) { sx = -sx; x2 = -1; }
    if (sy < 0) { sy = -sy; y2 = -1; }
    sx = max(sx, 1);

    if ((sy << 8) < 106 * sx) y2 = 0;
    if ((sy << 8) > 617 * sx) x2 = 0;

    int p1_local = ((ylocal+1 + y2) *(BLOCK_SIZE_X+2) ) + xlocal + x2 +1;
    int p2_local = ((ylocal+1 - y2) *(BLOCK_SIZE_X+2) ) + xlocal - x2 +1;
    int p3_local = ((ylocal+1 + y2) *(BLOCK_SIZE_X+2) ) + xlocal - x2 +1;
    int p4_local = ((ylocal+1 - y2) *(BLOCK_SIZE_X+2) ) + xlocal + x2 +1;

    int p0_global = (yglobal *w ) + xglobal;
    int p1_global = ((yglobal + y2) *w ) + xglobal + x2;
    int p2_global = ((yglobal - y2) *w ) + xglobal - x2;
    //    int p3_global = ((yglobal + y2) *w ) + xglobal - x2;
    //    int p4_global = ((yglobal - y2) *w ) + xglobal + x2;



    bool sn  = (LocalMemIx[p0_local] * LocalMemIx[p1_local] + LocalMemIy[p0_local] * LocalMemIy[p1_local]) > 0;
    bool ss  = (LocalMemIx[p0_local] * LocalMemIx[p2_local] + LocalMemIy[p0_local] * LocalMemIy[p2_local]) > 0;
    bool sew = (LocalMemIx[p3_local] * LocalMemIx[p4_local] + LocalMemIy[p3_local] * LocalMemIy[p4_local]) > 0;


    int m = 512;
    if (x2 != 0 && y2 != 0) m = 1024;
    if (!sn &&  ss & sew) m = 1024;
    if (            !sew) m = 1024;

    int m0 = LocalMemI[p0_local];
    int m1 = LocalMemI[p1_local];

    int m2 = LocalMemI[p2_local];
    int a0 = m0 * m;
    int a1 = m1 << 8;
    int a2 = m2 << 8;


    //ptBufferDxIn[p0_global] = p1_global;// p2_local>1156 | p2_local<0;//p1_global>307200;

    if (a1 > a0 && m1 >= m2) {

        atomicAdd(&ptSobelOut[p1_global], m0);
        atomicSub(&ptSobelOut[p0_global], m0);

    } else if (a2 > a0 && m2 >= m1) {

        atomicAdd(&ptSobelOut[p2_global], m0);
        atomicSub(&ptSobelOut[p0_global], m0);
    }

}


__global__ void Pass0_2(u_int32_t *ptSobelIn,
                        u_int32_t *ptLabelOut,
                        int w,int h)
{
    //__shared__ u_int32_t  LocalMemI[(BLOCK_SIZE_X+4)*(BLOCK_SIZE_Y+4)];

    int x = blockIdx.x*blockDim.x;
    int y = blockIdx.y*blockDim.y;
    int xglobal = x+threadIdx.x;
    int yglobal = y+threadIdx.y;
    int xlocal  = threadIdx.x;
    int ylocal  = threadIdx.y;


    /*for(int i=threadIdx.x  ;i<BLOCK_SIZE_X+4 && (x+i)<w;i=i+blockDim.x)
    {
        for(int j=threadIdx.y ;j<BLOCK_SIZE_Y+4 && (y+j)<h;j=j+blockDim.y)
        {
            if((x+i)<0 || (y+j)<0) continue;
            LocalMemI[i+j*(BLOCK_SIZE_X+4)]  = ptSobelIn[x+i-4 + (y+j-4)*w];
        }
    }

    syncthreads();
    */

    if(xglobal <= 4 || xglobal >= 640-4  || yglobal <= 4 || yglobal >= 480-4)
    {
        ptLabelOut[xglobal+yglobal*w] = 0;
        return;
    }

    int xglobal0 = xglobal;
    int yglobal0 = yglobal;
    int p_global  = xglobal+yglobal*w;
    int p_local  = xlocal+4+(ylocal+4)*(BLOCK_SIZE_X+4);
    int p0 = p_local, p1;


    //if (LocalMemI[p0] > 300000) {
    if (ptSobelIn [p_global] > 30000) {
        ptLabelOut[p_global] = p_global;
    }else
    {
        ptLabelOut[p_global] = 0;
    }

    return;


    /*
    if (LocalMemI[p0] < 3000) {
        ptLabelOut[p_global] = 0;
        return;
    }
    p1 = p0 - BLOCK_SIZE_Y+4;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        yglobal0--;
    }

    p1 = p0 - 1;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        xglobal0--;
    }

    p1 = p0 - BLOCK_SIZE_Y+4;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        yglobal0--;
    }

    p1 = p0 - 1;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        xglobal0--;
    }

    p1 = p0 - BLOCK_SIZE_Y+4;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        yglobal0--;
    }

    p1 = p0 - 1;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        xglobal0--;
    }

    p1 = p0 - BLOCK_SIZE_Y+4;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        yglobal0--;
    }

    p1 = p0 - 1;
    if (LocalMemI[p1] >= 3000) {
        p0 = p1;
        xglobal0--;
    }

    ptLabelOut[p_global] = xglobal0+yglobal0*w;
    */
}
